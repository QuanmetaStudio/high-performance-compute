#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
int Max = 8192;
int width = 32;

__global__ void Add(double *A, double *b, double *sum, const int Max)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < Max && j < Max)
        sum[i * Max + j] = A[i * Max + j] + b[i * Max + j];
}

double CPU_time(double *a, double *b, double *c)
{
    clock_t start = clock();
    for (int i = 0; i < Max * Max; ++i)
        c[i] = a[i] + b[i];
    clock_t end = clock();
    return (end - start) * 1000 / CLOCKS_PER_SEC;
}

void Check_result(double *a, double *b, double *sum)
{
    bool flag = true;
    for (int i = 0; i < Max; ++i)
        for (int j = 0; j < Max; ++j)
        {
            if (a[i * Max + j] + b[i * Max + j] != sum[i * Max + j])
            {
                flag = false;
                printf("Fail : (%d,%d)\n", i, j);
                printf("Correct Answer :%lf , My Answer :%lf\n",
                       (i - 0.1 * j + 1) + (0.2 * j - 0.1 * i), sum[i * Max + j]);
            }
        }
    if (flag == true)
        printf("GPU result True\n");
}

int main()
{
	clock_t st,en;
    
    double *a_h = (double *)malloc(Max * Max * sizeof(double));
    double *b_h = (double *)malloc(Max * Max * sizeof(double));
    double *c_h = (double *)malloc(Max * Max * sizeof(double));
    for (int i = 0; i < Max; ++i)
        for (int j = 0; j < Max; ++j)
        {
            a_h[i * Max + j] = i - 0.1 * j + 1;
            b_h[i * Max + j] = 0.2 * j - 0.1 * i;
            c_h[i * Max + j] = 0;
        }

    
    double *A, *B, *C;
    hipMalloc((void **)&A, Max * Max * sizeof(double));
    hipMalloc((void **)&B, Max *Max * sizeof(double));
    hipMalloc((void **)&C, Max * Max * sizeof(double));
	//start time
	st=clock();
	
    // Memcpy CPU -> GPU
    hipMemcpy(A, a_h, Max * Max * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(B, b_h, Max * Max * sizeof(double), hipMemcpyHostToDevice);

    
    dim3 block(width, width);
    dim3 grid(Max/block.x, Max/block.y);
    Add<<<grid, block>>>(A, B, C, Max);
	
   
    hipMemcpy(c_h, C,Max * Max * sizeof(double), hipMemcpyDeviceToHost);
	
	en=clock();
	float time1=(float)(en-st)*1000/CLOCKS_PER_SEC;
   
    
    Check_result(a_h, b_h, c_h);

    // print time of GPU,CPU

    printf("GPU Time:%lf ms \n", time1);
    printf("CPU Time:%lf ms \n", CPU_time(a_h, b_h, c_h));


    // free
   
    hipFree(A);
    hipFree(B);
    hipFree(C);
    free(a_h);
    free(b_h);
    free(c_h);
}