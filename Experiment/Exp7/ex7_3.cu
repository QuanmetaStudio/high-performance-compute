//使用constant memory存放向量
//global memory

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<time.h>
#include <stdlib.h>


int Max=16384;
int width=32;
double err = 0.1;

__constant__ double con_b[8192];

__global__ void multi(double *A,double *C,const int Max,int i){

	int idx=threadIdx.x+blockDim.x*blockIdx.x;
	//int idy=threadIdx.y+blockDim.y*blockIdx.y;
	if(idx<Max){
		int k=0;
		double sum=0;
		for(k=i*Max/2;k<(i+1)*Max/2;k++){
			sum+=A[idx*Max+k]*con_b[k%(Max/2)];
		}
		C[idx]+=sum;
	}
	
}

int main(){
	printf("使用constant memory存放向量:\n");
	double *A =(double *)malloc(Max * Max * sizeof(double));   //A	
	double b[Max];   //b
	double *C =(double *)malloc(Max * sizeof(double));   //C
	double *test_c=(double *)malloc(Max * sizeof(double));  //cpu_test
	
	int i,j;
	for(i=0;i<Max;i++){
		for(j=0;j<Max;j++){
			A[i*Max+j]=i-0.1*j+1;
		}
	}
	for(i=0;i<Max;i++){
		b[i]=log(sqrt(i*i-i+2));
		C[i]=0.0;
	}
	
	double *A_d,*C_d;
	hipMalloc((void **)&A_d,Max * Max * sizeof(double));
	
	hipMalloc((void **)&C_d,Max *sizeof(double));
	
	clock_t start,end;
	start=clock();
	hipMemcpy(A_d, A,Max*Max*sizeof(double),hipMemcpyHostToDevice);
	//cudaMemcpyToSymbol(con_b, b, sizeof(double) * Max);
	hipMemcpy(C_d, C,Max * sizeof(double), hipMemcpyHostToDevice);
	
	dim3 block(width,1);
	dim3 grid(Max/block.x, 1);
	for(int i=0;i<2;i++){
		hipMemcpyToSymbol(HIP_SYMBOL(con_b), &b[i*Max/2], sizeof(double) * Max/2);
		multi<<<grid,block>>>(A_d,C_d,Max,i);
		
	}
	
	hipMemcpy(C, C_d, Max * sizeof(double), hipMemcpyDeviceToHost);

	end=clock();
	double time=(end-start)*1000/CLOCKS_PER_SEC;
	
	//cpu:
    clock_t start_c,end_c;
    start_c=clock();
	
    for (int i = 0; i < Max; ++i){

        for (int j = 0; j < Max; ++j)
         {
             test_c[i]+=A[i*Max+j]*b[j];
         }
    }
    end_c=clock();
	double time_C=(end_c-start_c)*1000/CLOCKS_PER_SEC;
	printf("GPU TIME:%lf ms\n",time);
	printf("CPU TIME:%lf ms\n",time_C);
	//check result:
	
    bool flag = true;
    for (int i = 0; i < Max; ++i){

        double a=test_c[i];
        double b=C[i];
        if (abs(a-b)>err)
        {
			printf("cpu:%lf gpu:%lf\n",a,b);
            flag = false;
        }

    }
    if (flag == true)
        printf("result correct\n");
    else{
        printf("resul wrong\n");
    }
	
	
	hipFree(A_d);
	hipFree(C_d);

	free(A);
	free(test_c);
	free(C);
	
	
}
