//合并 访存

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<time.h>
#include <stdlib.h>

int Max=16384;
int width=32;

typedef struct  {
    double A1;
    double A2;
    double A3;
    double A4;
}stru;

__global__ void multi(stru *A,stru *b,double *C,const int Max){
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
	
    double sum=0.0;
    if(idx<Max && idy<Max && idx==idy){
        for(int i=0;i<Max/4;i++){

            sum+=A[idx*Max/4+i].A1*b[i*4].A1+A[idx*Max/4+i].A2*b[i*4+1].A1+A[idx*Max/4+i].A3*b[i*4+2].A1+A[idx*Max/4+i].A4*b[i*4+3].A1;
			
        }
        C[idx]=sum;
    }
	
}

int main(){
	printf("合并访存:\n");
	stru *A =(stru *)malloc(Max * Max/4 * sizeof(stru));   //A	
	stru *b =(stru *)malloc(Max * sizeof(stru));   //b
	double *C =(double *)malloc(Max * sizeof(double));   //C
	double *test_c=(double *)malloc(Max * sizeof(double));  //cpu_test
	
	int i,j;
	for(i=0;i<Max;i++){
		for(j=0;j<Max/4;j++){
			A[i*Max/4+j].A1=i-0.1*j*4+1;
			A[i*Max/4+j].A2=i-(0.1*j*4+1)+1;
			A[i*Max/4+j].A3=i-(0.1*j*4+2)+1;
			A[i*Max/4+j].A4=i-(0.1*j*4+3)+1;
		}
	}
	for(i=0;i<Max;i++){
		b[i].A1=log(sqrt(i*i-i+2));
        b[i].A2=0.0;
        b[i].A3=0.0;
        b[i].A4=0.0;
		C[i]=0.0;
	}
	
	stru *A_d,*b_d;
	double *C_d;
	hipMalloc((void **)&A_d,Max * Max/4 * sizeof(stru));
	hipMalloc((void **)&b_d,Max *sizeof(stru));
	hipMalloc((void **)&C_d,Max *sizeof(double));
	
	clock_t start,end;
	start=clock();
	hipMemcpy(A_d, A,Max*Max/4*sizeof(stru),hipMemcpyHostToDevice);
	hipMemcpy(b_d, b,Max*sizeof(stru),hipMemcpyHostToDevice);
	hipMemcpy(C_d, C,Max * sizeof(double), hipMemcpyHostToDevice);
	
	dim3 block(width,width);
	dim3 grid(Max/block.x, Max/block.y);
	multi<<<grid,block>>>(A_d,b_d,C_d,Max);
	
	hipMemcpy(C, C_d, Max * sizeof(double), hipMemcpyDeviceToHost);

	end=clock();
	double time=(end-start)*1000/CLOCKS_PER_SEC;
	
	//check result:
	
    
	//cpu:
    clock_t start_c,end_c;
    start_c=clock();
	
    for (int i = 0; i < Max; ++i){

        for (int j = 0; j < Max/4; ++j)
         {
             test_c[i]+=A[i*Max/4+j].A1*b[j*4].A1+A[i*Max/4+j].A2*b[j*4+1].A1+A[i*Max/4+j].A3*b[j*4+2].A1+A[i*Max/4+j].A4*b[j*4+3].A1;
         }
    }
    end_c=clock();
	bool flag = true;
    for (int i = 0; i < Max; ++i){

        float a=test_c[i];
        float b=C[i];
        if (a!=b)
        {
			printf("cpu:%lf gpu:%lf\n",a,b);
            flag = false;
        }

    }
    if (flag == true)
        printf("result correct\n");
    else{
        printf("resul wrong\n");
    }
	
	double time_C=(end_c-start_c)*1000/CLOCKS_PER_SEC;
	printf("GPU TIME:%lf ms\n",time);
	printf("CPU TIME:%lf ms\n",time_C);
	
	hipFree(A_d);
	hipFree(b_d);
	hipFree(C_d);

	free(A);
	free(b);
	free(C);
	
	
}
