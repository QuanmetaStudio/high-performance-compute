//global memory

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<time.h>
#include <stdlib.h>

int Max=16384;
int width=32;

__global__ void multi(double *A,double *b,double *C,const int Max){
	int idx=threadIdx.x+blockDim.x*blockIdx.x;
	int idy=threadIdx.y+blockDim.y*blockIdx.y;
	if(idx<Max && idy<Max && idx==idy){
		int k=0;
		double sum=0;
		for(k=0;k<Max;k++){
			sum+=A[idx*Max+k]*b[k];
		}
		C[idx]=sum;
	}
	
}

int main(){
	printf("global memory:\n");
	double *A =(double *)malloc(Max * Max * sizeof(double));   //A	
	double *b =(double *)malloc(Max * sizeof(double));   //b
	double *C =(double *)malloc(Max * sizeof(double));   //C
	double *test_c=(double *)malloc(Max * sizeof(double));  //cpu_test
	
	int i,j;
	for(i=0;i<Max;i++){
		for(j=0;j<Max;j++){
			A[i*Max+j]=i-0.1*j+1;
		}
	}
	for(i=0;i<Max;i++){
		b[i]=log(sqrt(i*i-i+2));
		C[i]=0.0;
	}
	
	double *A_d,*b_d,*C_d;
	hipMalloc((void **)&A_d,Max * Max * sizeof(double));
	hipMalloc((void **)&b_d,Max *sizeof(double));
	hipMalloc((void **)&C_d,Max *sizeof(double));
	
	clock_t start,end;
	start=clock();
	hipMemcpy(A_d, A,Max*Max*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(b_d, b,Max*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(C_d, C,Max * sizeof(double), hipMemcpyHostToDevice);
	
	dim3 block(width,width);
	dim3 grid(Max/block.x, Max/block.y);
	multi<<<grid,block>>>(A_d,b_d,C_d,Max);
	
	hipMemcpy(C, C_d, Max * sizeof(double), hipMemcpyDeviceToHost);

	end=clock();
	double time=(end-start)*1000/CLOCKS_PER_SEC;
	
	//cpu:
    clock_t start_c,end_c;
    start_c=clock();
	
    for (int i = 0; i < Max; ++i){

        for (int j = 0; j < Max; ++j)
         {
            test_c[i]+=A[i*Max+j]*b[j];
         }
    }
    end_c=clock();
	double time_C=(end_c-start_c)*1000/CLOCKS_PER_SEC;
	printf("GPU TIME:%lf ms\n",time);
	printf("CPU TIME:%lf ms\n",time_C);
	
	//check result:
	
    bool flag = true;
    for (int i = 0; i < Max; ++i){

        float a=test_c[i];
        float b=C[i];
        if (a!=b)
        {
            flag = false;
        }

    }
    if (flag == true)
        printf("result correct\n");
    else{
        printf("resul wrong\n");
    }
	
	hipFree(A_d);
	hipFree(b_d);
	hipFree(C_d);

	free(A);
	free(b);
	free(C);
	
	
}
