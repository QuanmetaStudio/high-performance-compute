// 使用global memory合并访存



#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<time.h>
#include <stdlib.h>
#define TILE_WIDTH 32

__global__ void multi(int m, int n, int k, double *A, double *B, double *C)
{
    int idx = blockIdx.y * blockDim.y + threadIdx.y;
    int idy	= blockIdx.x * blockDim.x + threadIdx.x;

    if ((idx < m) && (idy < k))
    {
        double Cvalue = 0.0;
        for (int i = 0; i < n; ++i)
            Cvalue += A[idx * n + i] * B[idy + i * k];
        C[idx * k + idy] = Cvalue;
    }
}


int main()
{
 
	printf("global memory:\n");
    int m = 4096, n = 4096, k = 4096;
    double *A = (double *)malloc(m * n * sizeof(double));
    double *B = (double *)malloc(n * k * sizeof(double));
    double *C = (double *)malloc(m * k * sizeof(double));
    double *test_c = (double *)malloc(m * k * sizeof(double));
	int i,j;
    for (i = 0; i < m; ++i)
        for (j = 0; j < m; ++j)
        {
            A[i * m + j] = (i - 0.1 * j + 1) / (i + j + 1);
            B[i * m + j] = (j - 0.2 * i + 1) * (i + j + 1) / (i * i + j * j + 1);
            C[i * m + j] = 0.0;
        }


    int size = sizeof(double);
 
	double *A_d,*b_d,*C_d;

    hipMalloc((void **)&A_d, m * n * size);
    hipMalloc((void **)&b_d, n * k * size);
    hipMalloc((void **)&C_d, m * k * size);
	
	clock_t start,end;
	start=clock();
    
    hipMemcpy(A_d, A, size * m * n, hipMemcpyHostToDevice);
    hipMemcpy(b_d, B, size * n * k, hipMemcpyHostToDevice);
    hipMemcpy(C_d, C, size * m * k, hipMemcpyHostToDevice);

 
    dim3 dimGrid((k - 1) / TILE_WIDTH + 1, (m - 1) / TILE_WIDTH + 1, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

   
    multi<<<dimGrid, dimBlock>>>(m, n, k, A_d, b_d, C_d);

    hipMemcpy(C, C_d, size * m * k, hipMemcpyDeviceToHost);

    end=clock();
	double time=(end-start)*1000/CLOCKS_PER_SEC;
	
	//cpu:
    clock_t start_c,end_c;
    start_c=clock();
	
    for (i = 0; i < m; ++i){

        for (j = 0; j < m; ++j)
         {
			double sum=0;
			for(int k=0;k<m;k++){
			sum+=A[i*m+k]*B[k*m+j];
			}
			test_c[i * m + j] = sum;
         }
    }
    end_c=clock();
	double time_C=(end_c-start_c)*1000/CLOCKS_PER_SEC;
	printf("GPU TIME:%lf ms\n",time);
	printf("CPU TIME:%lf ms\n",time_C);

    //比较结果
    bool flag = true;
    for (int i = 0; i < 10; ++i)
    {
        if (abs(test_c[i] - C[i]) > 0.001)
        {
            flag = false;
			//printf("cpu:%lf gpu:%lf\n",test_c[i],C[i]);
            
        }
    }
    if (flag)
		printf("Result Correct!\n");
    else
		printf("Result Error!\n");
        

    hipFree(A_d);
    hipFree(b_d);
    hipFree(C_d);
    free(A);
    free(B);
    free(C);
    free(test_c);
    return 0;
}