//采用分块乘法，使用shared memory


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<time.h>
#include <stdlib.h>
#define TILE_WIDTH 16

__global__ void multi(int m, int n, int k, double *A, double *B, double *C)
{
    
    __shared__ double S_a[TILE_WIDTH][TILE_WIDTH];
    __shared__ double S_b[TILE_WIDTH][TILE_WIDTH];

    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    
    int idx = by * TILE_WIDTH + ty;
    int idy = bx * TILE_WIDTH + tx;

    
    double flag = 0;

    
    for (int t = 0; t < (n - 1) / TILE_WIDTH + 1; ++t)
    {
        
        if (idx < m && t * TILE_WIDTH + tx < n)              
                                                             
            S_a[tx][ty] = A[idx * n + t * TILE_WIDTH + tx]; 
        else
            S_a[tx][ty] = 0.0;

        if (t * TILE_WIDTH + ty < n && idy < k)
           
            S_b[tx][ty] = B[(t * TILE_WIDTH + ty) * k + idy];
        else
            S_b[tx][ty] = 0.0;

       
        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; ++i)
            flag += S_a[i][ty] * S_b[tx][i]; 

        __syncthreads();

        if (idx < m && idy < k)
            C[idx * k + idy] = flag;
    }
}

int main()
{
    printf("shared memory:\n");
    int m = 4096, n = 4096, k = 4096;
    double *A = (double *)malloc(m * n * sizeof(double));
    double *B = (double *)malloc(n * k * sizeof(double));
    double *C = (double *)malloc(m * k * sizeof(double));
    double *test_c = (double *)malloc(m * k * sizeof(double));
	int i,j;
    for (i = 0; i < m; ++i)
        for (j = 0; j < m; ++j)
        {
            A[i * m + j] = (i - 0.1 * j + 1) / (i + j + 1);
            B[i * m + j] = (j - 0.2 * i + 1) * (i + j + 1) / (i * i + j * j + 1);
            C[i * m + j] = 0.0;
        }

    
    int size = sizeof(double);
 
	double *A_d,*b_d,*C_d;
    hipMalloc((void **)&A_d, m * n * size);
    hipMalloc((void **)&b_d, n * k * size);
    hipMalloc((void **)&C_d, m * k * size);

    clock_t start,end;
	start=clock();

    hipMemcpy(A_d, A, size * m * n, hipMemcpyHostToDevice);
    hipMemcpy(b_d, B, size * n * k, hipMemcpyHostToDevice);
    hipMemcpy(C_d, C, size * m * k, hipMemcpyHostToDevice);

   
    dim3 dimGrid((k - 1) / TILE_WIDTH + 1, (m - 1) / TILE_WIDTH + 1, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);

    
    multi<<<dimGrid, dimBlock>>>(m, n, k, A_d, b_d, C_d);

    
    hipMemcpy(C, C_d, size * m * k, hipMemcpyDeviceToHost);
	end=clock();
	double time=(end-start)*1000/CLOCKS_PER_SEC;
    
	//cpu:
    clock_t start_c,end_c;
    start_c=clock();
	
    for (i = 0; i < m; ++i){

        for (j = 0; j < m; ++j)
         {
			double sum=0;
			for(int k=0;k<m;k++){
			sum+=A[i*m+k]*B[k*m+j];
			}
			test_c[i * m + j] = sum;
         }
    }
    end_c=clock();
	double time_C=(end_c-start_c)*1000/CLOCKS_PER_SEC;
	printf("GPU TIME:%lf ms\n",time);
	printf("CPU TIME:%lf ms\n",time_C);
    

    //比较结果
    bool flag = true;
    for (int i = 0; i < m * k; ++i)
    {
        if (abs(test_c[i] - C[i]) > 0.001)
        {
            flag = false;
            //printf("cpu:%lf gpu:%lf\n",test_c[i],C[i]);
        }
    }
    if (flag)
		printf("Result Correct!\n");
    else
		printf("Result Error!\n");


    hipFree(A_d);
    hipFree(b_d);
    hipFree(C_d);
	
    free(A);
    free(B);
    free(C);
    free(test_c);
    return 0;
}